/*
Copyright (c) 2019 SChernykh

This file is part of RandomX CUDA.

RandomX CUDA is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

RandomX CUDA is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with RandomX CUDA.  If not, see<http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "../CryptoNight_constants.h"
#include "../cryptonight.h"
#include "../cuda_device.hpp"
#include "../workers/Workers.h"

void randomx_prepare(nvid_ctx *ctx, const uint8_t* seed_hash, xmrig::Variant variant, uint32_t batch_size)
{
    randomx_dataset* dataset = Workers::getDataset(seed_hash, variant);
    const size_t dataset_size = randomx_dataset_item_count() * RANDOMX_DATASET_ITEM_SIZE;
    if (!ctx->d_rx_dataset) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_dataset, dataset_size));
    }
    if (!ctx->d_long_state) {
        ctx->d_scratchpads_size = batch_size * rx_select_memory(variant);
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_long_state, ctx->d_scratchpads_size));
    }
    if (!ctx->d_rx_hashes) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_hashes, batch_size * 64));
    }
    if (!ctx->d_rx_entropy) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_entropy, batch_size * (128 + 2560)));
    }
    if (!ctx->d_rx_vm_states) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_vm_states, batch_size * 2560));
    }
    if (!ctx->d_rx_rounding) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_rounding, batch_size * sizeof(uint32_t)));
    }

    if ((memcmp(ctx->rx_dataset_seedhash, seed_hash, sizeof(ctx->rx_dataset_seedhash)) != 0) || (ctx->rx_variant != variant)) {
        memcpy(ctx->rx_dataset_seedhash, seed_hash, sizeof(ctx->rx_dataset_seedhash));
        ctx->rx_variant = variant;
        CUDA_CHECK(ctx->device_id, hipMemcpy(ctx->d_rx_dataset, randomx_get_dataset_memory(dataset), dataset_size, hipMemcpyHostToDevice));
    }
}
